#include <iostream>
#include <cassert>
#include <functional>
#include "../matrix.hpp"

using namespace std;

int main() {
  matrix<int> m1 = matrix<int>(2, 2);
  m1.set(0, 0, 1);
  m1.set(0, 1, 2);
  m1.set(1, 0, 3);
  m1.set(1, 1, 4);
  matrix<int> m2 = m1 * m1;
  cout << "Test 1: Multiplication" << endl;
  assert(m2[0][0] == 7);
  assert(m2[1][1] == 22);


  cout << "Test 2: Addition" << endl;
  m2 = m1 + m1;
  assert(m2[0][0] == 2);
  assert(m2[1][1] == 8);


  cout << "Test 3: Subtraction" << endl;
  matrix<int> m1Neg = -m1;
  m2 = m2 + m1Neg;
  assert(m2[0][0] == m1[0][0]);
  assert(m2[1][1] == m1[1][1]);


  cout << "Test 4: Transpose" << endl;
  matrix<float> m8(10, 20);
  for (int i = 0; i < 10; ++i) {
    for (int j = 0; j < 20; ++j) {
      m8.set(i, j, i * 20 + j);
    }
  }
  matrix<float> m8trans = m8.transpose();
  for (int i = 0; i < 10; ++i) {
    for (int j = 0; j < 20; ++j) {
      assert(m8[i][j] == m8trans[j][i]);
    }
  }
  m2 = m1.transpose();
  assert(m2[1][0] == m1[0][1]);
  assert(m2[1][1] == m1[1][1]);
 
  cout << "Test 5: CUDA Multiplication" << endl;
  matrix<int> m3(20, 30);
  for (int i = 0; i < 20; ++i) {
    for (int j = 0; j < 30; ++j) {
      m3.set(i, j, 1);
    }
  }
  matrix<int> m4(30, 40);
  for (int i = 0; i < 30; ++i) {
    for (int j = 0; j < 40; ++j) {
      m4.set(i, j, 1);
    }
  }
  matrix<int> m5 = m3 * m4;
  assert(m5[0][0] == 30 && m5[19][39] == 30);

  cout << "Test 6: CUDA Addition" << endl;
  matrix<int> m6 = m4 + m4;
  assert(m6[29][39] == 2);

  cout << "Test 7: CUDA Scaling" << endl;
  matrix<int> m7 = m4 * 2;
  assert(m7[29][39] == 2);

  cout << "All tests passed" << endl;
}